#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, German Research Center for Artificial Intelligence (DFKI)
 * Author: Clemens Lutz <clemens.lutz@dfki.de>
 *
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the <organization> nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// If defined, use CUDA managed memory, otherwise allocate with malloc
#define USE_MANAGED

// If defined, then set the memAdviseSetAccessedBy flag
// #define ADVISE_ACCESSED_BY

// If defined, then set the memAdviseSetReadMostly flag
// #define ADVISE_READ_MOSTLY

// #define ADVISE_PREFERRED_LOCATION_CPU

// If defined, then touch the data on the host between kernel launches to avoid
// device-side caching
// #define TOUCH_ON_HOST

// If defined, then read data on GPU, else write data on GPU
#define OP_READ

// 32 GiB of data
constexpr unsigned long long SIZE = 32 * 1024 * 1024 * (1024 / sizeof(int));

// Prefetch data in 16 MiB blocks
constexpr unsigned long long PREFETCH_SIZE = 16 * 1024 * 1024 / sizeof(int);

// Number of runs
constexpr unsigned RUNS = 5;

// Device
constexpr int DEVICE_ID = 0;

// NUMA node
constexpr int NUMA_NODE = 0;

#ifndef USE_MANAGED
#include <cstdlib>
#endif

#include <algorithm>
#include <chrono>
#include <iostream>
#include <utility>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <numa.h>

#define CHECK_CUDA(ans) check_cuda((ans), __FILE__, __LINE__)
void check_cuda(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
      std::cerr
          << "Exit with code "
          << hipGetErrorString(code)
          << " (" << code << ") "
          << "in file " << file << ":" << line
          << std::endl;
      std::exit(1);
    }
}

__global__ void read_kernel(int *data, uint64_t len, int *result) {
    const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned gstride = blockDim.x * gridDim.x;

    int counter = 0;
    for (uint64_t i = gid; i < len; i += gstride) {
        counter += data[i];
    }

    atomicAdd(result, counter);
}

__global__ void write_kernel(int *data, uint64_t len) {
    const unsigned gid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned gstride = blockDim.x * gridDim.x;

    for (uint64_t i = gid; i < len; i += gstride) {
        data[i] = i;
    }
}

int main() {
    // Kernel launch parameters
    int sm_count = 0;
    CHECK_CUDA(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, DEVICE_ID));
    int warp_size = 0;
    CHECK_CUDA(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, DEVICE_ID));
    const unsigned GRID_DIM = sm_count * 2;
    const unsigned BLOCK_DIM = warp_size * 4;

    std::cout
        << "Running on device " << DEVICE_ID
        << " with grid dim " << GRID_DIM
        << " and block dim " << BLOCK_DIM
        << " and prefetching " << SIZE / PREFETCH_SIZE << " data blocks"
        << std::endl;

    // Set CUDA device
    CHECK_CUDA(hipSetDevice(DEVICE_ID));

    // Set NUMA node
    numa_run_on_node(NUMA_NODE);

    // Allocate managed memory
    int *data = nullptr;
#ifdef USE_MANAGED
    CHECK_CUDA(hipMallocManaged(&data, SIZE * sizeof(int)));
    std::cout << "Managed memory enabled" << std::endl;

#ifdef ADVISE_READ_MOSTLY
    CHECK_CUDA(hipMemAdvise(data, SIZE * sizeof(int), hipMemAdviseSetReadMostly, DEVICE_ID));
    std::cout << "hipMemAdviseSetReadMostly enabled" << std::endl;
#endif

#ifdef ADVISE_ACCESSED_BY
    CHECK_CUDA(hipMemAdvise(data, SIZE * sizeof(int), hipMemAdviseSetAccessedBy, DEVICE_ID));
    std::cout << "hipMemAdviseSetAccessedBy enabled" << std::endl;
#endif

#ifdef ADVISE_PREFERRED_LOCATION_CPU
    CHECK_CUDA(hipMemAdvise(data, SIZE * sizeof(int), hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    std::cout << "hipMemAdviseSetPreferredLocation CPU enabled" << std::endl;
#endif

#else
    data = (int*) numa_alloc_onnode(SIZE * sizeof(int), NUMA_NODE);
    std::cout << "System memory enabled" << std::endl;
#endif

#ifdef TOUCH_ON_HOST
    std::cout << "Touch on host between runs enabled" << std::endl;
#endif

    // Fill data array
    for (uint64_t i = 0; i < SIZE; ++i) {
        data[i] = i;
    }

    // Allocate result
    int *result = nullptr;
    CHECK_CUDA(hipMalloc(&result, sizeof(int)));

    // Setup events
    hipEvent_t start_timer[2], end_timer[2], e1, e2, et;
    CHECK_CUDA(hipEventCreate(&start_timer[0]));
    CHECK_CUDA(hipEventCreate(&start_timer[1]));
    CHECK_CUDA(hipEventCreate(&end_timer[0]));
    CHECK_CUDA(hipEventCreate(&end_timer[1]));
    CHECK_CUDA(hipEventCreate(&e1));
    CHECK_CUDA(hipEventCreate(&e2));
    CHECK_CUDA(hipEventCreate(&et));

    // Setup streams
    hipStream_t s1, s2, s3, st;
    CHECK_CUDA(hipStreamCreateWithFlags(&s1, hipStreamNonBlocking));
    CHECK_CUDA(hipStreamCreateWithFlags(&s2, hipStreamNonBlocking));
    CHECK_CUDA(hipStreamCreateWithFlags(&s3, hipStreamNonBlocking));

#ifdef OP_READ
    std::cout << "Running read kernel" << std::endl;
#else
    std::cout << "Running write kernel" << std::endl;
#endif

    uint64_t num_tiles = SIZE / PREFETCH_SIZE;
    for (unsigned run = 0; run < RUNS; ++run) {
        std::chrono::steady_clock::time_point timer_start = std::chrono::steady_clock::now();

        // prefetch first tile
        hipMemPrefetchAsync(data, PREFETCH_SIZE * sizeof(int), DEVICE_ID, s2);
        hipEventRecord(e1, s2); 

        for (uint64_t i = 0; i < num_tiles; i++) { 
            // make sure previous kernel and current tile copy both completed 
            hipEventSynchronize(e1);  
            hipEventSynchronize(e2);

            // run multiple kernels on current tile 
            read_kernel<<<GRID_DIM, BLOCK_DIM, 0, s1>>>(&data[i * PREFETCH_SIZE], PREFETCH_SIZE, result);
            hipEventRecord(e1, s1); 

            // prefetch next tile to the gpu in a separate stream 
            if (i < num_tiles-1) {
                // make sure the stream is idle to force non-deferred HtoD prefetches first 
                hipStreamSynchronize(s2);       
                hipMemPrefetchAsync(&data[(i + 1) * PREFETCH_SIZE], PREFETCH_SIZE * sizeof(int), DEVICE_ID, s2); 
                hipEventRecord(e2, s2); 
            } 

            // offload current tile to the cpu after the kernel is completed using the deferred path 
            /* hipMemPrefetchAsync(a + tile_size * i, tile_size * sizeof(size_t), hipCpuDeviceId, s1);  */

            // rotate streams and swap events 
            st = s1; s1 = s2; s2 = st; 
            st = s2; s2 = s3; s3 = st; 
            et = e1; e1 = e2; e2 = et; 
        }

        // Wait for kernel completion
        CHECK_CUDA(hipDeviceSynchronize());

        std::chrono::steady_clock::time_point timer_end = std::chrono::steady_clock::now();
        std::chrono::milliseconds time_span = std::chrono::duration_cast<std::chrono::milliseconds>(timer_end - timer_start);
        double time_ms = time_span.count();

        // Compute and print throughput in GiB/s
        uint64_t size_GiB = (SIZE * sizeof(int)) / 1024 / 1024 / 1024;
        double tput = ((double)size_GiB) / time_ms * 1000.0;
        std::cout << "Throughput: " << tput << " GiB/s" << std::endl;

#ifdef TOUCH_ON_HOST
        for (uint64_t i = 0; i < SIZE; ++i) {
            data[i] = run + i;
        }
#endif
    }

    // Cleanup
    CHECK_CUDA(hipStreamDestroy(s1));
    CHECK_CUDA(hipStreamDestroy(s2));
    CHECK_CUDA(hipStreamDestroy(s3));
    CHECK_CUDA(hipEventDestroy(start_timer[0]));
    CHECK_CUDA(hipEventDestroy(start_timer[1]));
    CHECK_CUDA(hipEventDestroy(end_timer[0]));
    CHECK_CUDA(hipEventDestroy(end_timer[1]));
#ifdef USE_MANAGED
    CHECK_CUDA(hipFree(data));
#else
    numa_free(data, SIZE * sizeof(int));
#endif
    CHECK_CUDA(hipFree(result));
}
